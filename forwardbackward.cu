#include "hip/hip_runtime.h"
#define MIN -1e36
#define MAXMIN -1e30

//kernel set
//(Maxlable_length,1) (batch,2)
__global__ void gpu_forward_backward(float *Output, float *Forward, float *Backward, int *Lable,
	int *Lable_Length, int *Length, int NodeSize, int OutLength, int FDLength, int MaxLable_Length)
{
	int bi = blockIdx.x;
	int bj = blockIdx.y;

	int ti = threadIdx.x;
	//int tj = threadIdx.y;

//the label with blank is no more than 128
	__shared__ float form[128];
	__shared__ float pr[128];

	//the start of batch
	int st = bi*MaxLable_Length;
	int l_l = Lable_Length[bi];
	int l = Length[bi];
	int fst = bi*FDLength;
	int ost = bi*OutLength;

	float cur;
	int label;
	bool mark;
	//load label
	if (ti < l_l)
		label = Lable[st + ti];

	__syncthreads();
	if (bj == 0 && ti < l_l)//forward compute
	{
		//mark compute way
		if (ti > 1 && label != Lable[st + ti - 2])
			mark = true;
		else
			mark = false;

		//initial the first time
		if (ti < 2)
		{
			form[ti] = logf(Output[ost + label]);
			Forward[fst + ti] = form[ti];
		}
		else
		{
			form[ti] = MIN;
			Forward[fst + ti] = 1;
		}
		__syncthreads();

		//loop the form time = 1 to time = T
		for (int t = 1; t < l; t++)
		{
			//load output
			pr[ti] = Output[ost + t*NodeSize + label];//at the time of t
			//compute log value
			if (pr[ti] == 0)
				pr[ti] = MIN;
			else
				pr[ti] = logf(pr[ti]);

			//if the rest time length < the label length 
			if (ti < (l_l - 2 * (l - t)))//斜率是 2 因此下边成立
				cur = MIN;
			else
			{
				//the first label
				if (ti == 0)
					cur = pr[ti] + form[ti]; //logf(Output[ost + t*NodeSize + lable[ti]]) + form[ti];
				else
				{
					//compute the value
					if (mark)
						cur = pr[ti] + //logf(Output[ost + t*NodeSize + lable[ti]])
						+Active::LogExchangeFunction(form[ti - 2],
						Active::LogExchangeFunction(form[ti - 1], form[ti]));
					else
						cur = pr[ti] + //logf(Output[ost + t*NodeSize + lable[ti]])
						+Active::LogExchangeFunction(form[ti - 1], form[ti]);
				}
			}
			//
			__syncthreads();
			//if the value is small 
			if (cur > MAXMIN)
			{
				Forward[fst + t*l_l + ti] = cur;
				form[ti] = cur;
			}
			else
			{
				Forward[fst + t*l_l + ti] = 1;
				form[ti] = MIN;
			}

			__syncthreads();
		}
	}
	else if (ti < l_l)//backward
	{
		//mark compute the no blank and no repet
		if (ti < l_l - 2 && label != Lable[st + ti + 2])
			mark = true;
		else
			mark = false;
		//initial the last time
		if (ti >= l_l - 2)
		{
			form[ti] = 0;
			Backward[fst + (l - 1)*l_l + ti] = 0;
		}
		else
		{
			form[ti] = MIN;
			Backward[fst + (l - 1)*l_l + ti] = 1;
		}

		__syncthreads();

		//loop from time T-1 to the first
		for (int t = l - 2; t >= 0; t--)
		{
			//load output
			pr[ti] = Output[ost + (t + 1)*NodeSize + label];//the next time

			if (pr[ti] == 0)
				pr[ti] = MIN;
			else
				pr[ti] = logf(pr[ti]);
			__syncthreads();

			if (ti > 2 * t + 1)
				cur = MIN;
			else
			{
				if (ti == l_l - 1)
					cur = pr[ti] +
					form[ti];
				else
				{
					if (mark)
						cur = Active::LogExchangeFunction(Active::LogExchangeFunction(
						pr[ti + 1] + form[ti + 1],
						pr[ti + 2] + form[ti + 2]),
						pr[ti] + form[ti]);
					else
						cur = Active::LogExchangeFunction(
						pr[ti + 1] + form[ti + 1],
						pr[ti] + form[ti]);
				}
			}
			__syncthreads();

			if (cur > MAXMIN)
			{
				Backward[fst + t*l_l + ti] = cur;
				form[ti] = cur;
			}
			else
			{
				Backward[fst + t*l_l + ti] = 1;
				form[ti] = MIN;
			}

			__syncthreads();
		}
	}
}
